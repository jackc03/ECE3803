#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "util.h"

#define EPSILON 0.00001f

// amount of floating point numbers between answer and computed value 
// for the answer to be taken correctly
const int maxUlps = 1000;

event_pair timer;
  
float4 host_force_calc(float4 A, float4 B) {
  float x = B.x - A.x;
  float y = B.y - A.y;
  float z = B.z - A.z;
  float rsq = x*x + y*y + z*z;
  // avoid divide by zero
  if(rsq < EPSILON) {
    rsq += EPSILON;
  }
  float r = sqrt(rsq); // distance between particles

  // Coulomb's law
  float f = A.w * B.w / rsq;
  float inv_r = 1.0f / r;

  // force vector has a magnitude of 1 and points along the axis of the charge
  float4 fv = make_float4(x*inv_r, y*inv_r, z*inv_r, f);
  return fv;
}
 
void host_force_eval(float4 *set_A, float4 *set_B, int *indices, float4 *force_vectors, int array_length) {
  for(int i=0; i < array_length; i++) {
    if(indices[i] < array_length && indices[i] >= 0) {
      force_vectors[i] = host_force_calc(set_A[i], set_B[indices[i]]);
    } else {
      force_vectors[i] = make_float4(0.0,0.0,0.0,0.0);
    }
  }
}

__device__ float4 device_calc(float4 A, float4 B) {
  float x = B.x - A.x;
  float y = B.y - A.y;
  float z = B.z - A.z;
  float rsq = x*x + y*y + z*z;

  // avoid divide by zero
  if(rsq < EPSILON) {
    rsq += EPSILON;
  }
  float r = sqrt(rsq); // distance between particles

  // Coulomb's law
  float f = A.w * B.w / rsq;
  float inv_r = 1.0f / r;

  // force vector has a magnitude of 1 and points along the axis of the charge
  float4 fv = make_float4(x*inv_r, y*inv_r, z*inv_r, f);
  return fv;
}


__global__ void force_eval(float4 *set_A, float4 *set_B, int *indices, float4 *force_vectors, int array_length) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if(indices[i] < array_length && indices[i] >= 0) {
    force_vectors[i] = device_calc(set_A[i],  set_B[indices[i]]);
  } else {
    force_vectors[i] = make_float4(0.0,0.0,0.0,0.0);
  }
}


void charged_particles(float4 *h_set_A, float4 *h_set_B, int *h_indices, float4 *h_force_vectors, int num_elements) { 
  // Declare pointers for all the cudaMallocs
  float4 *set_A, *set_B, *force_vectors;
  int *indices;


  // Malloc each pointer
  size_t size = num_elements * sizeof(float4);
  hipMalloc(&force_vectors, size);
  hipMalloc(&set_A,         size);
  hipMalloc(&set_B,         size);
  hipMalloc(&indices,       num_elements * sizeof(int));


  // Now copy memory over
  hipMemcpy(force_vectors, h_force_vectors, size, hipMemcpyHostToDevice);
  hipMemcpy(set_A, h_set_A, size, hipMemcpyHostToDevice);
  hipMemcpy(set_B, h_set_B, size, hipMemcpyHostToDevice);
  hipMemcpy(indices, h_indices, num_elements * sizeof(int), hipMemcpyHostToDevice);
  

  // Stealing parameters from part 1
  int block_size = 1024;
  int grid_size = (num_elements + block_size - 1) / block_size;

  
  // Launch kernels
  start_timer(&timer);
  force_eval<<<grid_size, block_size>>>(set_A, set_B, indices, force_vectors,  num_elements);


  check_launch("gpu force eval");
  stop_timer(&timer,"gpu force eval");
  

  // Now copy memory over
  hipMemcpy(h_force_vectors, force_vectors, size, hipMemcpyDeviceToHost);
  hipMemcpy(h_set_A, set_A, size, hipMemcpyDeviceToHost);
  hipMemcpy(h_set_B, set_B, size, hipMemcpyDeviceToHost);
  hipMemcpy(h_indices, indices, num_elements * sizeof(int), hipMemcpyDeviceToHost);

  hipFree(force_vectors);
  hipFree(set_A);
  hipFree(set_B);
  hipFree(indices);
}


int main(void) {
  // create arrays of 4M elements
  int num_elements =  1 << 22;

  // pointers to host & device arrays
  float4 *h_set_A = 0;
  float4 *h_set_B = 0;
  int *h_indices = 0;
  float4 *h_force_vectors = 0;
  float4 *h_force_vectors_checker = 0;
  
   // initialize
  srand(time(NULL)); 
  
  // malloc host array
  h_set_A = (float4*)malloc(num_elements * sizeof(float4));
  h_set_B = (float4*)malloc(num_elements * sizeof(float4));
  h_indices = (int*)malloc(num_elements * sizeof(int));
  h_force_vectors = (float4*)malloc(num_elements * sizeof(float4));
  h_force_vectors_checker = (float4*)malloc(num_elements * sizeof(float4));
  
  // if either memory allocation failed, report an error message
  if(h_set_A == 0 || h_set_B == 0 || h_force_vectors == 0 || h_indices == 0 || h_force_vectors_checker == 0) {
    printf("couldn't allocate memory\n");
    exit(1);
  }

  // generate random input
  for(int i=0; i < num_elements; i++) {
    h_set_A[i] = make_float4(rand(), rand(), rand(), rand()); 
    h_set_B[i] = make_float4(rand(), rand(), rand(), rand());

    // some indices will be invalid
    h_indices[i] = rand() % (num_elements + 2);
  }
  
  start_timer(&timer);
  // generate reference output
  host_force_eval(h_set_A, h_set_B, h_indices, h_force_vectors_checker, num_elements);
  check_launch("host force eval");
  stop_timer(&timer, "host force eval");
  
  // the results of the calculation need to end up in h_force_vectors;
  charged_particles(h_set_A, h_set_B, h_indices, h_force_vectors, num_elements);
  
  // check CUDA output versus reference output
  int error = 0;
  
  for(int i=0; i<num_elements; i++) {
    float4 v = h_force_vectors[i];
    float4 vc = h_force_vectors_checker[i];

    if( !AlmostEqual2sComplement(v.x,vc.x,maxUlps) ||
      !AlmostEqual2sComplement(v.y,vc.y,maxUlps) ||
      !AlmostEqual2sComplement(v.z,vc.z,maxUlps) ||
      !AlmostEqual2sComplement(v.w,vc.w,maxUlps)) { 
      error = 1;
    }
  }
  
  if(error) {
    printf("Failure: Output of CUDA version and CPU version didn't match\n");
  }
  else {
    printf("Success: CUDA and reference output match\n");
  }
 
 
  // deallocate memory
  free(h_set_A);
  free(h_set_B);
  free(h_indices);
  free(h_force_vectors);
  free(h_force_vectors_checker);
}

